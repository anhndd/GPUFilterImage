#include "hip/hip_runtime.h"
#include "string.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <stdlib.h>
extern "C" {
#include "ppm_lib.h"
}
#define MAX_RUN_TIME 1

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d \n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
#define FILTER_SIZE 25

int filterSofter[FILTER_SIZE] = {0, 0, 0, 0, 0, 0, 1, 3, 1, 0, 0, 3, 5,
                                 3, 0, 0, 1, 3, 1, 0, 0, 0, 0, 0, 0};

int filterSharpen[FILTER_SIZE] = {-1, -1, -1, -1, -1, -1, -1, -1, -1,
                                  -1, -1, -1, 49, -1, -1, -1, -1, -1,
                                  -1, -1, -1, -1, -1, -1, -1};
int filterBlur[FILTER_SIZE] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 2, 3,
                               2, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
int filterHoriSobel[FILTER_SIZE] = {1,  2,  0,  -2, -1,  4,  8, 0, -8,
                                    -4, 6,  12, 0,  -12, -6, 4, 8, 0,
                                    -8, -4, 1,  2,  0,   -2, -1};
int filterShatter[FILTER_SIZE] = {1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0,
                                  0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1};

int filterSoften[FILTER_SIZE] = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};

int filterVerSobel[FILTER_SIZE] = {-1, -4, -6, -4, -1, -2, -8, -12, -8,
                                   -2, 0,  0,  0,  0,  0,  2,  8,   12,
                                   8,  2,  1,  4,  6,  4,  1};

int calculDivisionFactor(int *filter, int sizeFilter) {
  int sum = 0;
  for (int i = 0; i < sizeFilter; i++) {
    sum += filter[i];
  }

  if (sum == 0)
    return 1;

  return sum;
}

__global__ void applyFilterGPUAvance(PPMPixel *img, int *filter, PPMPixel *data,
                                     int divisionFactor) {
  int gridCounter = 0;
  int finalRed = 0;
  int finalBlue = 0;
  int finalGreen = 0;
  for (int y2 = -2; y2 <= 2; y2++) {
    for (int x2 = -2; x2 <= 2; x2++) {
      int Y = blockIdx.x + y2;
      int X = threadIdx.x + x2;
      if (X > -1 && Y > -1 && X < blockDim.x && Y < gridDim.x) {
        int position = X + Y * blockDim.x;
        finalRed += img[position].red * filter[gridCounter];
        finalBlue += img[position].blue * filter[gridCounter];
        finalGreen += img[position].green * filter[gridCounter];
      }
      gridCounter++;
    }
  }
  int positionPixel = threadIdx.x + blockIdx.x * blockDim.x;
  finalRed /= divisionFactor;
  finalBlue /= divisionFactor;
  finalGreen /= divisionFactor;
  data[positionPixel].red = finalRed;
  data[positionPixel].blue = finalBlue;
  data[positionPixel].green = finalGreen;
}

int main() {
  hipEvent_t start, stop;
  float time;
  PPMImage *image;
  char name[100];
  int *filter;

  int typeOutput = 0;
  char nameFile[100];
  printf("input type: ");
  scanf("%d", &typeOutput);

  while (typeOutput != -1) {
    double gpu_time_used = 0;
    image = readPPM("mon_image.ppm");
    switch (typeOutput) {
    case 0:
      strcpy(name, "mon_image_dest_softer.ppm");
      strcpy(nameFile, "result/main_b_softer.txt");
      filter = filterSofter;
      break;
    case 1:
      strcpy(name, "mon_image_dest_soften.ppm");
      strcpy(nameFile, "result/main_b_soften.txt");
      filter = filterSoften;
      break;
    case 2:
      strcpy(name, "mon_image_dest_sharpen.ppm");
      strcpy(nameFile, "result/main_b_sharpen.txt");
      filter = filterSharpen;
      break;
    case 3:
      strcpy(name, "mon_image_dest_shatter.ppm");
      strcpy(nameFile, "result/main_b_shatter.txt");
      filter = filterShatter;
      break;
    case 4:
      strcpy(name, "mon_image_dest_blur.ppm");
      strcpy(nameFile, "result/main_b_blur.txt");
      filter = filterBlur;
      break;
    case 5:
      strcpy(name, "mon_image_dest_horisobel.ppm");
      strcpy(nameFile, "result/main_b_horisobel.txt");
      filter = filterHoriSobel;
      break;
    case 6:
      strcpy(name, "mon_image_dest_versobel.ppm");
      strcpy(nameFile, "result/main_b_versobel.txt");
      filter = filterVerSobel;
      break;
    default:
      break;
    }

    // changeColorPPM(image);
    writePPM(name, image);
    //     PPMImage *imgDestination = readPPM(name);

    PPMPixel *dev_dataImage;
    PPMPixel *dev_dataDestination;
    int *dev_filter;

    // start setup data malloc
    HANDLE_ERROR(hipMalloc((PPMPixel **)&dev_dataImage,
                            image->x * image->y * sizeof(PPMPixel)));
    HANDLE_ERROR(hipMemcpy(dev_dataImage, image->data,
                            image->x * image->y * sizeof(PPMPixel),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((PPMPixel **)&dev_dataDestination,
                            image->x * image->y * sizeof(PPMPixel)));
    HANDLE_ERROR(hipMemcpy(dev_dataDestination, image->data,
                            image->x * image->y * sizeof(PPMPixel),
                            hipMemcpyHostToDevice));

    hipMalloc((int **)&dev_filter, FILTER_SIZE * sizeof(int));
    hipMemcpy(dev_filter, filter, FILTER_SIZE * sizeof(int),
               hipMemcpyHostToDevice);
    // end setup data malloc

    int divisionFactor = calculDivisionFactor(filter, FILTER_SIZE);

    int loop = 0;
    double arrayTime[MAX_RUN_TIME + 1];
    while (loop < MAX_RUN_TIME) {
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start, 0);
      applyFilterGPUAvance<<<image->y, image->x>>>(
          dev_dataImage, dev_filter, dev_dataDestination, divisionFactor);
      hipDeviceSynchronize();
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);
      arrayTime[loop] = time;
      loop++;
    }

    HANDLE_ERROR(hipMemcpy(image->data, dev_dataDestination,
                            image->x * image->y * sizeof(PPMPixel),
                            hipMemcpyDeviceToHost));
    printf(">%s \n", hipGetErrorString(hipGetLastError()));
    writePPM(name, image);

    // write result to file
    FILE *f = fopen(nameFile, "wb");
    for (int i = 0; i < MAX_RUN_TIME; i++) {
      gpu_time_used += arrayTime[i];
      fprintf(f, "%f\n", arrayTime[i]);
    }
    arrayTime[MAX_RUN_TIME] = gpu_time_used;
    fprintf(f, "%f\n", arrayTime[MAX_RUN_TIME]);
    fclose(f);
    printf("apply successful: %3.5f ms\n", gpu_time_used);
    printf("%d %d\n", image->x, image->y);

    // free cuda
    hipFree(dev_dataImage);
    hipFree(dev_filter);

    typeOutput++;
    if (typeOutput >= 7) {
      break;
    }
    // printf("input type: ");
    // scanf("%d", &typeOutput);
  }
  free(image);
}